#include <iostream>
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <stdio.h>

#define FFTCompute _declspec(dllexport)
#define MyFunc _declspec(dllexport)

extern "C" {
	MyFunc int addNums(int a, int b) {
		return a + b;
	}

	FFTCompute float** computeFFT(float *numbers, int size) {

		size_t memorySize = size * sizeof(hipfftComplex);
		int complexItemNumbers = size / 2 + 1;

		float* device_input;
		hipfftComplex* host_output;
		hipfftComplex* device_output;
		hipfftHandle plan;

		//allocate host memory for output
		host_output = (hipfftComplex*)malloc((size / 2 + 1) * sizeof(hipfftComplex));
		//allocate memory for final output;

		//allocate device memory for input
		hipMalloc(&device_input, size * sizeof(float));
		//allocate device memory for output
		hipMalloc(&device_output, (size/2 + 1) * sizeof(hipfftComplex));
		// creating 1D plan
		hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
		// copying numbers array to device memory
		hipMemcpy(device_input, numbers,
			size * sizeof(float), hipMemcpyHostToDevice);
		// run fowards fft, real to complex
		hipfftExecR2C(plan, device_input, device_output);
		//copying output from device to host
		hipMemcpy(host_output, device_output, 
			(size/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

		// copying output to 2D array to return
		float** output = new float*[complexItemNumbers];
		for (int i = 0; i < complexItemNumbers; i++) {
			output[i] = new float[2];
		}

		for (int i = 0; i < complexItemNumbers; i++) {
			output[i][0] = host_output[i].x;
			output[i][1] = host_output[i].y;
		}

		return output;
	}
}